#include "hip/hip_runtime.h"
#include <cstdio>
#include <spdlog/spdlog.h>
#include <hip/hip_runtime.h>

#include "check.hpp"

hipStream_t stream1, stream2, stream3;
size_t nbytes = 1024 * 1024 * 1024;
float *da, *db;
float *ha, *hb;

std::mutex mtx;
std::condition_variable cv;
bool flag = false;

__global__ void infinite_loop_kernel() {
  while (true);
}

void proxy_func() {
  //CHECK_CUDA(hipSetDevice(0));

  hipCtx_t ctx_proxy;

  spdlog::info("proxy_func: before hipCtxCreate");
  CHECK_CU(hipCtxCreate(&ctx_proxy, hipDeviceScheduleSpin | hipDeviceMapHost, 0));
  spdlog::info("proxy_func: after hipCtxCreate");

  spdlog::info("proxy_func: before hipCtxSetCurrent");
  CHECK_CU(hipCtxSetCurrent(ctx_proxy));
  spdlog::info("proxy_func: after hipCtxSetCurrent");

  spdlog::info("proxy_func: now waiting...");
  {
    std::unique_lock<std::mutex> lk(mtx);
    cv.wait(lk, []{ return flag; });
  }

  std::this_thread::sleep_for(std::chrono::milliseconds(1000));

  spdlog::info("proxy_func: before hipMemcpyAsync");
  //CHECK_CUDA(hipMemcpyAsync(db, hb, nbytes, hipMemcpyDefault, stream3));
  float *tmp;
  CHECK_CUDA(hipMalloc(&tmp, nbytes));
  spdlog::info("proxy_func: after hipMemcpyAsync");
}

//void test_ctx() {
//  int signature = 0xdeadbeef;
//  CHECK_CUDA(hipMalloc(&da, nbytes));
//  CHECK_CUDA(hipMemcpy(da, &signature, sizeof(int), hipMemcpyDefault));
//  hipCtxCreate(&ctx_proxy, hipDeviceScheduleSpin, 0);
//  hipCtxSetCurrent(ctx_proxy);
//  CHECK_CUDA(hipMalloc(&db, nbytes));
//  CHECK_CUDA(hipMemcpy(db, da, nbytes, hipMemcpyDefault));
//  int new_signature;
//  CHECK_CUDA(hipMemcpy(&new_signature, db, sizeof(int), hipMemcpyDefault));
//  printf("%x\n", new_signature);
//}

int main() {
  //test_ctx();

  CHECK_CUDA(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));
  //CHECK_CUDA(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));
  stream2 = 0;
  CHECK_CUDA(hipStreamCreateWithFlags(&stream3, hipStreamNonBlocking));

  auto proxy_thread = std::thread(proxy_func);

  hipEvent_t ev;
  CHECK_CUDA(hipEventCreate(&ev));

  CHECK_CUDA(hipMalloc(&da, nbytes));
  CHECK_CUDA(hipHostMalloc(&ha, nbytes));
  CHECK_CUDA(hipMalloc(&db, nbytes));
  CHECK_CUDA(hipHostMalloc(&hb, nbytes));

  infinite_loop_kernel<<<1, 1, 0, stream1>>>();
  CHECK_CUDA(hipEventRecord(ev, stream1));
  CHECK_CUDA(hipStreamWaitEvent(stream2, ev, 0));

  {
    std::unique_lock<std::mutex> lk(mtx);
    flag = true;
    cv.notify_one();
  }

  spdlog::info("main: before hipMemcpyAsync");
  CHECK_CUDA(hipMemcpy(ha, da, nbytes, hipMemcpyDefault));
  spdlog::info("main: after hipMemcpyAsync");

  proxy_thread.join();
  
  return 0;
}