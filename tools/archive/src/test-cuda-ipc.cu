#include <cstdio>
#include <spdlog/spdlog.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <mpi.h>

#include "check.hpp"

int num_gpus = 4;
hipStream_t stream1, stream2;
size_t nbytes = 1024 * 1024 * 1024;
float *da, *db, *dc;
float *ha, *hb;
hipIpcMemHandle_t da_handle;

std::mutex mtx;
std::condition_variable cv;
int flag = 0;

void thread1_func() {
  CHECK_CUDA(hipMalloc(&db, nbytes));
  hb = (float*)malloc(nbytes);

  {
    std::unique_lock<std::mutex> lk(mtx);
    flag += 1;
    cv.notify_all();
  }

  {
    std::unique_lock<std::mutex> lk(mtx);
    cv.wait(lk, []{ return flag == 2; });
  }

  for (int i = 0; i < 100; ++i) {
    spdlog::info("thread1: {}", i);
    CHECK_CUDA(hipMemcpy(hb, db, nbytes, hipMemcpyDefault));
  }
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipGetLastError());
}

void thread2_func() {
  hipCtx_t ctx_proxy;
  CHECK_CU(hipCtxCreate(&ctx_proxy, hipDeviceScheduleSpin | hipDeviceMapHost, 0));
  CHECK_CU(hipCtxSetCurrent(ctx_proxy));

  CHECK_CUDA(hipIpcOpenMemHandle((void**)&da, da_handle, hipIpcMemLazyEnablePeerAccess));

  CHECK_CUDA(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));
  CHECK_CUDA(hipMalloc(&dc, nbytes));

  {
    std::unique_lock<std::mutex> lk(mtx);
    flag += 1;
    cv.notify_all();
  }

  {
    std::unique_lock<std::mutex> lk(mtx);
    cv.wait(lk, []{ return flag == 2; });
  }

  std::this_thread::sleep_for(std::chrono::milliseconds(1000));

  for (int i = 0; i < 100; ++i) {
    spdlog::info("thread2: {}", i);
    CHECK_CUDA(hipMemcpyAsync(dc, da, nbytes, hipMemcpyDefault, stream2));
  }
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipGetLastError());
}

int main() {
  CHECK_MPI(MPI_Init(nullptr, nullptr));

  int rank, size;
  CHECK_MPI(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
  CHECK_MPI(MPI_Comm_size(MPI_COMM_WORLD, &size));

  if (rank == 0) {
    CHECK_CUDA(hipSetDevice(1));
    CHECK_CUDA(hipMalloc(&da, nbytes));
    CHECK_CUDA(hipIpcGetMemHandle(&da_handle, da));
    CHECK_MPI(MPI_Send(&da_handle, sizeof(hipIpcMemHandle_t), MPI_BYTE, 1, 0, MPI_COMM_WORLD));
  } else if (rank == 1) {
    CHECK_MPI(MPI_Recv(&da_handle, sizeof(hipIpcMemHandle_t), MPI_BYTE, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE));

    auto thread1 = std::thread(thread1_func);
    auto thread2 = std::thread(thread2_func);

    thread1.join();
    thread2.join();
  }

  CHECK_MPI(MPI_Finalize());

  return 0;
}