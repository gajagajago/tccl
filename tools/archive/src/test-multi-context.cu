#include <cstdio>
#include <spdlog/spdlog.h>
#include <hip/hip_runtime.h>
#include <chrono>

#include "check.hpp"

int num_gpus = 4;
hipStream_t stream1, stream2;
size_t nbytes = 1024 * 1024 * 1024;
float *da, *db, *d1a;
float *ha, *hb;
hipIpcMemHandle_t d1a_handle;

std::mutex mtx;
std::condition_variable cv;
int flag = 0;

void thread1_func() {
  //hipCtx_t ctx_proxy;
  //CHECK_CU(hipCtxCreate(&ctx_proxy, hipDeviceScheduleSpin | hipDeviceMapHost, 0));
  //CHECK_CU(hipCtxSetCurrent(ctx_proxy));
  //CHECK_CUDA(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));
  CHECK_CUDA(hipMalloc(&da, nbytes));
  //CHECK_CUDA(hipHostMalloc(&ha, nbytes));
  ha = (float*)malloc(nbytes);

  {
    std::unique_lock<std::mutex> lk(mtx);
    flag += 1;
    cv.notify_all();
  }

  {
    std::unique_lock<std::mutex> lk(mtx);
    cv.wait(lk, []{ return flag == 2; });
  }

  for (int i = 0; i < 100; ++i) {
    spdlog::info("thread1: {}", i);
    CHECK_CUDA(hipMemcpy(ha, da, nbytes, hipMemcpyDefault));
  }
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipGetLastError());
}

void thread2_func() {
  hipCtx_t ctx_proxy;
  CHECK_CU(hipCtxCreate(&ctx_proxy, hipDeviceScheduleSpin | hipDeviceMapHost, 0));
  CHECK_CU(hipCtxSetCurrent(ctx_proxy));

  //float *d1a_0;
  //CHECK_CUDA(hipIpcOpenMemHandle((void**)&d1a_0, d1a_handle, hipIpcMemLazyEnablePeerAccess));

  CHECK_CUDA(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));
  CHECK_CUDA(hipHostMalloc(&hb, nbytes));
  CHECK_CUDA(hipMalloc(&db, nbytes));

  {
    std::unique_lock<std::mutex> lk(mtx);
    flag += 1;
    cv.notify_all();
  }

  {
    std::unique_lock<std::mutex> lk(mtx);
    cv.wait(lk, []{ return flag == 2; });
  }

  std::this_thread::sleep_for(std::chrono::milliseconds(1000));

  for (int i = 0; i < 100; ++i) {
    spdlog::info("thread2: {}", i);
    void* tmp;
    CHECK_CUDA(hipMallocAsync(&tmp, 1024, stream2));
    //CHECK_CUDA(hipMemcpyAsync(db, hb, nbytes, hipMemcpyDefault, stream2));
    //if (i % 2 == 0) {
    //  CHECK_CUDA(hipMemcpyAsync(db, hb, nbytes, hipMemcpyDefault, hipStreamPerThread));
    //} else {
    //  CHECK_CUDA(hipMemcpyAsync(db, hb, nbytes, hipMemcpyDefault, hipStreamLegacy));
    //}
  }
  CHECK_CUDA(hipDeviceSynchronize());
  CHECK_CUDA(hipGetLastError());
}

int main() {
  CHECK_CUDA(hipSetDevice(1));
  CHECK_CUDA(hipMalloc(&d1a, nbytes));
  CHECK_CUDA(hipIpcGetMemHandle(&d1a_handle, d1a));

  //CHECK_CUDA(hipSetDevice(0));
  //float *d1a_0;
  //CHECK_CUDA(hipIpcOpenMemHandle((void**)&d1a_0, d1a_handle, hipIpcMemLazyEnablePeerAccess));

  CHECK_CUDA(hipSetDevice(0));

  auto thread1 = std::thread(thread1_func);
  auto thread2 = std::thread(thread2_func);

  //{
  //  std::unique_lock<std::mutex> lk(mtx);
  //  flag = true;
  //  cv.notify_all();
  //}

  thread1.join();
  thread2.join();

  return 0;
}